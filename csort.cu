#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

extern "C" {
  void sort_int_wrapper( int *data, int N)
  {
    thrust::device_ptr<int> dev_ptr(data);
    thrust::sort(dev_ptr, dev_ptr+N);
  }
  void sort_double_wrapper( double *data, int N)
  {
    thrust::device_ptr<double> dev_ptr(data);
    thrust::sort(dev_ptr, dev_ptr+N);
  }
  void sort_by_key_int_wrapper( int *key, int N, int *data)
  {
    thrust::device_ptr<int> dev_key(key);
    thrust::device_ptr<int> dev_ptr(data);
    thrust::stable_sort_by_key(dev_key, dev_key+N, dev_ptr);
  }
  void sort_by_key_double_wrapper( double *key, int N, int *data)
  {
    thrust::device_ptr<double> dev_key(key);
    thrust::device_ptr<int> dev_ptr(data);
    thrust::stable_sort_by_key(dev_key, dev_key+N, dev_ptr);
  }
  void prefixSum_wrapper( int *data, int *out ,int N)
  {
    thrust::device_ptr<int> dev_data(data);
    thrust::device_ptr<int> dev_out(out);
    thrust::exclusive_scan(dev_data, dev_data+N, dev_out, 1);
  }
}
